#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>


// CUDA kernel for vector addition
__global__ void vectorAdd(const float *a, const float *b, float *c, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    int n = 1000000; // Size of the vectors
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_a = new float[n];
    float *h_b = new float[n];
    float *h_c = new float[n];

    // Initialize host vectors
    for (int i = 0; i < n; ++i)
    {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc((void **)&d_a, size));
    CUDA_CHECK(hipMalloc((void **)&d_b, size));
    CUDA_CHECK(hipMalloc((void **)&d_c, size));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());

    // Copy the result from device to host
    CUDA_CHECK(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    // Verify the result
    for (int i = 0; i < 10; ++i) // Check the first 10 results
    {
        std::cout << "h_a[" << i << "] + h_b[" << i << "] = " << h_c[i] << std::endl;
    }

    // Free device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}

